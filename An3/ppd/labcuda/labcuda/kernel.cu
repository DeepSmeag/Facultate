#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""
#include <time.h>
#include <stdio.h>

#define MATRIX_SIZE 10
#define MASK_SIZE 3
#define TILE_WIDTH 10
#define NO_THREADS 5

__global__ void convolutionKernel(int* matrix, int* convolution)
{
    // Shared memory to store the matrix tile
    __shared__ int conv[MASK_SIZE][MASK_SIZE];
    int temp[3][MATRIX_SIZE];
    // Temporary vectors for convolution
    int startRow = threadIdx.x * (MATRIX_SIZE / NO_THREADS);
    int endRow = startRow + MATRIX_SIZE / NO_THREADS - 1;
    for (int i = 0; i < MASK_SIZE * MASK_SIZE; i++)
    {
        conv[i / MASK_SIZE][i % MASK_SIZE] = convolution[i];
    }
    for (int i = 0; i < MASK_SIZE; i++)
    {
        temp[0][i] = matrix[(startRow - 1 >= 0 ? startRow - 1 : 0) * MATRIX_SIZE + i];
        temp[1][i] = matrix[startRow * MATRIX_SIZE + i];
        temp[2][i] = matrix[(endRow + 1 < MATRIX_SIZE ? endRow + 1 : MATRIX_SIZE - 1) * MATRIX_SIZE + i];

	}
    __syncthreads();

    bool up = true;

    for (int i = startRow; i < endRow; i++)
    {
        if (up)
        {
            // using temp[0] us the "above" vector and temp[1] as the "current" vector
            for (int j = 0; j < MATRIX_SIZE; j++)
            {
                temp[1][j] = matrix[i * MATRIX_SIZE + j];
                // doing the convolution row by row
                int sum = 0;
                // first row
                for (int m = 0; m < 3; m++)
                {
                    int correctCol = (j + m - 1) < 0 ? j : (j + m - 1);
                    correctCol = (correctCol < MATRIX_SIZE ? correctCol : MATRIX_SIZE - 1);
                    sum += temp[0][correctCol] * conv[0][m];
                    sum += temp[1][correctCol] * conv[1][m];
                    sum += matrix[(i+1) * MATRIX_SIZE + correctCol] * conv[2][m];
                }
                matrix[i * MATRIX_SIZE + j] = sum;
            }
        }
        else
        {
            // using temp[0] us the "current" vector and temp[1] as the "above" vector
            for (int j = 0; j < MATRIX_SIZE; j++)
            {
                temp[0][j] = matrix[i * MATRIX_SIZE + j];
                // doing the convolution row by row
                int sum = 0;
                // first row
                for (int m = 0; m < 3; m++)
                {
                    int correctCol = (j + m - 1) < 0 ? j : (j + m - 1);
                    correctCol = (correctCol < MATRIX_SIZE ? correctCol : MATRIX_SIZE - 1);
                    sum += temp[1][correctCol] * conv[0][m];
                    sum += temp[0][correctCol] * conv[1][m];
                    sum += matrix[(i + 1) * MATRIX_SIZE + correctCol] * conv[2][m];
                }
                matrix[i * MATRIX_SIZE + j] = sum;
            }
        }

        up = !up;
    }
    // Synchronize to make sure all threads have finished convolution
    __syncthreads();
    /*if (!up && threadIdx.x == 0) {
        printf("Thread %d\n", threadIdx.x);
        for (int j = 0; j < MATRIX_SIZE; j++) {
            printf("%d ", temp[1][j]);
        }
        printf("\n");
        for (int j = 0; j < MATRIX_SIZE; j++) {
            printf("%d ", temp[0][j]);
        }
        printf("-------------\n");
    }*/
    // doing the last row, that is end - 1
    for (int j = 0; j < MATRIX_SIZE; j++)
    {
        //temp[up? 1 : 0][j] = matrix[endRow * MATRIX_SIZE + j];
        // same process, we know up's value
        int sum = 0;
        for (int m = 0; m < 3; m++)
        {
            int correctCol = (j + m - 1) < 0 ? j : (j + m - 1);
            correctCol = (correctCol < MATRIX_SIZE ? correctCol : MATRIX_SIZE - 1);
            sum += temp[up ? 0 : 1][correctCol] * conv[0][m];
            //sum += temp[up ? 1 : 0][correctCol] * conv[1][m];
            sum += matrix[endRow * MATRIX_SIZE + correctCol] * conv[1][m];
            sum += temp[2][correctCol] * conv[2][m];
        }
        matrix[endRow * MATRIX_SIZE + j] = sum;
    }
   

    
}

int main()
{
    const int matrixSize = MATRIX_SIZE;
    const int matrixTotalSize = matrixSize * matrixSize;
    int matrix[matrixTotalSize];
    int convolution[9] = { 0, 0, 0, 0, 2, 0, 0, 0, 0 };
    clock_t start_time = clock();
    // Initialize matrix with values for testing
    for (int i = 0; i < matrixTotalSize; ++i) {
        matrix[i] = i;
    }

    int* dev_matrix = 0;
    int* dev_conv = 0;
    hipMalloc((void**)&dev_matrix, matrixTotalSize * sizeof(int));
    hipMalloc((void**)&dev_conv, 3 * 3 * sizeof(int));
    hipMemcpy(dev_matrix, matrix, matrixTotalSize * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_conv, convolution, 3 * 3 * sizeof(int), hipMemcpyHostToDevice);

    // Launch the convolution kernel with 1 block of 10 threads
    convolutionKernel << <1, NO_THREADS >> > (dev_matrix, dev_conv);
    hipDeviceSynchronize();

    // Copy the result back to the host
    hipMemcpy(matrix, dev_matrix, matrixTotalSize * sizeof(int), hipMemcpyDeviceToHost);
    clock_t end_time = clock();
    double elapsed_time = ((double)(end_time - start_time)) / CLOCKS_PER_SEC;

    // Print the elapsed time
    printf("Elapsed time: %f seconds\n", elapsed_time);
    // Print the result
    for (int i = 0; i < matrixSize; ++i) {
        for (int j = 0; j < matrixSize; ++j) {
            printf("%d\t", matrix[i * matrixSize + j]);
        }
        printf("\n");
    }

    hipFree(dev_matrix);

    return 0;
}
