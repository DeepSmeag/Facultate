#include "hip/hip_runtime.h"
#include <stdio.h>

__global__ matProd() {
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    printf("Hello World from GPU! Row: %d\n", row);
}

int main() {
    printf("Hello World from CPU!\n");

    matProd <<<2, 20000 >>>();
    hipDeviceSynchronize();
    return 0;
}
